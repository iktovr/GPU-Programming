#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <vector>
#include <tuple>
#include <iomanip>
#include <cmath>
#include <string>

#define STBI
#ifdef STBI
#include "../common/stb_image_include.hpp"
#endif

#define TIME
#ifdef TIME
#include <chrono>
using namespace std::chrono;
#endif

#ifndef __HIPCC__
#define __host__
#define __device__
#endif

#include "../common/vec3.hpp"

#include "primitives.hpp"
#include "scene.hpp"
#include "render.hpp"
#include "ssaa.hpp"

int main() {
	// TODO: argv
	bool gpu = true;

	vec3 Ka(0.2), Kd(0.7), Ks(0.7);
	double p = 100;

	int frames;
	std::string path;
	int width, height;
	Camera camera;

	Material cube_material{vec3(0), Ka, Kd, Ks, p, 0.0, 0.0}, octahedron_material{vec3(0), Ka, Kd, Ks, p, 0.0, 0.0}, icosahedron_material{vec3(0), Ka, Kd, Ks, p, 0.0, 0.0};
	vec3 cube_origin, octahedron_origin, icosahedron_origin;
	double cube_scale, octahedron_scale, icosahedron_scale;
	int cube_lights, octahedron_lights, icosahedron_lights;

	Mesh floor(
		{{{5, 5, 0}, {0, 0, 1}}, {{-5, -5, 0}, {0, 0, 1}}, {{5, -5, 0}, {0, 0, 1}}, {{-5, 5, 0}, {0, 0, 1}}},
		{{0, 1, 2, 0}, {0, 3, 1, 0}}
	);
	std::string texture_path;
	Material floor_material{vec3(0), Ka, Kd, Ks, p, 0.0, 0.0};

	int lights_count;
	std::vector<Light> lights;

	int max_depth, ssaa_coeff;

	std::cin >> frames >> path >> width >> height >> camera
	         >> cube_origin >> cube_material.color >> cube_scale >> cube_material.reflection >> cube_material.refraction >> cube_lights 
	         >> octahedron_origin >> octahedron_material.color >> octahedron_scale >> octahedron_material.reflection >> octahedron_material.refraction >> octahedron_lights 
	         >> icosahedron_origin >> icosahedron_material.color >> icosahedron_scale >> icosahedron_material.reflection >> icosahedron_material.refraction >> icosahedron_lights 
	         >> floor.vertexes[0].point >> floor.vertexes[1].point >> floor.vertexes[2].point >> floor.vertexes[3].point
	         >> texture_path >> floor_material.color >> floor_material.reflection;

	std::cin >> lights_count;
	for (int i = 0; i < lights_count; ++i) {
		vec3 pos, color;
		std::cin >> pos >> color;
		lights.push_back({pos, color});
	}

	std::cin >> max_depth >> ssaa_coeff;

	Scene scene;
	scene.add_material({{0, 0, 0.6}, {0.2, 0.2, 0.2}, {0.7, 0.7, 0.7}, {0.7, 0.7, 0.7}, 100, 0, 0});
	scene.add_material(cube_material);
	scene.add_material(floor_material);

	int edge_mtl = 0, 
	    cube_mtl = 1,
	    floor_mtl = 2;

	scene.add_mesh(floor, {floor_mtl});
	Mesh cube("objects/cube.obj");
	scene.add_mesh(cube, {cube_mtl, edge_mtl}, cube_origin, cube_scale);

	scene.add_light(lights);
	scene.ambient_light = {1, 1, 1};

	// std::cout << scene;
	// return 0;

	RawScene raw_scene;
	if (gpu) {
		raw_scene = scene.get_gpu_raw_scene();
	} else {
		raw_scene = scene.get_raw_scene();
	}
	
	char buff[512];
	std::vector<vec3f> frame(width * height * ssaa_coeff * ssaa_coeff);
	std::vector<vec3c> data(width * height);

#ifdef TIME
	double frame_time = 0;
#endif

	double t;
	int field = std::floor(std::log10(frames - 1)) + 1;
	for(int k = 0; k < frames; k++) {
		t = 2 * PI / frames * k;
		camera.at(t);

#ifdef TIME
		steady_clock::time_point start = steady_clock::now();
#endif
		if (!gpu) {
			cpu::render(raw_scene, camera, frame, width * ssaa_coeff, height * ssaa_coeff, max_depth);
		} else {
			gpu::render(raw_scene, camera, frame, width * ssaa_coeff, height * ssaa_coeff, max_depth);
		}

#ifdef TIME
		steady_clock::time_point end = steady_clock::now();
		frame_time += duration_cast<nanoseconds>(end - start).count() / 1000000.0;
#endif

		// std::sprintf(buff, path.c_str(), field, k);
		std::cerr << "\rFrames remaining: " << std::setw(field) << std::setfill(' ') << (frames - k - 1);
		
		// ssaa(frame, data, width, height, ssaa_coeff);

		for (int i = 0; i < frame.size(); ++i) {
			data[i].x = (unsigned char)(std::min(frame[i].x, 1.0f) * 255);
			data[i].y = (unsigned char)(std::min(frame[i].y, 1.0f) * 255);
			data[i].z = (unsigned char)(std::min(frame[i].z, 1.0f) * 255);
		} 

		stbi_write_png(path.c_str(), width, height, 3, data.data(), width * 3);

		// std::ofstream out_file(path, std::ios::binary);
		// check(out_file.is_open(), false, "failed to open output file");

		// out_file.write(reinterpret_cast<char*>(&width), sizeof(width));
		// out_file.write(reinterpret_cast<char*>(&height), sizeof(height));
		// out_file.write(reinterpret_cast<char*>(data.data()), sizeof(vec3c) * width * height);
	}
	std::cerr << "\nConverting to gif...\n";
	// std::system("convert res/*.png res.gif");

#ifdef TIME
	frame_time /= frames;
	std::cout << "Frame time: " << frame_time << '\n';
#endif

	raw_scene.clear();

	return 0;
}