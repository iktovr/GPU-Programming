#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "../common/error_checkers.hpp"

__global__ void reduce(int* idata, int n, int* odata) {
	int tid = threadIdx.x;
	int id = blockDim.x * blockIdx.x * 2 + threadIdx.x;

	extern __shared__ int sdata[];

	sdata[tid] = idata[id] + idata[id + blockDim.x];
	__syncthreads();
	for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		odata[blockIdx.x] = sdata[0];
	}
}

const size_t BLOCK_SIZE = 1024;

template <class T>
inline T get_block_count(T size, T block_size, T log_block_size) {
	return (size >> (log_block_size + 1)) + ((size & ((block_size << 1) - 1)) > 0);
}

template <class T>
T ceil_2_pow(T a) {
	while ((a & (a - 1)) != 0) {
		a &= a - 1;
	}
	return a << 1;
}

template <class T>
T log2(T a) {
	T log = 0;
	while (a > 1) {
		++log;
		a >>= 1;
	}
	return log;
}

int reduce(const std::vector<int>& data) {
	size_t data_size = ceil_2_pow(data.size());
	std::vector<int> fill(data_size - data.size(), 0);
	size_t log_block_size = log2(BLOCK_SIZE);
	size_t res_size = get_block_count(data_size, BLOCK_SIZE, log_block_size);

	int *dev_data, *dev_res;
	cudaCheck(hipMalloc(&dev_data, sizeof(int) * data_size));
	cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(int) * data.size(), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(dev_data + data.size(), fill.data(), sizeof(int) * fill.size(), hipMemcpyHostToDevice));
	cudaCheck(hipMalloc(&dev_res, sizeof(int) * res_size));

	while (res_size > 1) {
		// std::cout << data_size << ' ' << res_size << '\n';
		reduce<<<res_size, BLOCK_SIZE, sizeof(int) * BLOCK_SIZE>>>(dev_data, data_size, dev_res);
		cudaCheck(hipDeviceSynchronize());
		cudaCheckLastError();
		std::swap(dev_data, dev_res);
		data_size = res_size;
		res_size = get_block_count(data_size, BLOCK_SIZE, log_block_size);
	}

	// std::cout << data_size << ' ' << res_size << '\n';
	reduce<<<1, (data_size >> 1), sizeof(int) * (data_size >> 1)>>>(dev_data, data_size, dev_res);
	cudaCheck(hipDeviceSynchronize());
	cudaCheckLastError();

	int res;
	cudaCheck(hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipFree(dev_data));
	cudaCheck(hipFree(dev_res));
	return res;
}

int main() {
	int n;
	std::cin >> n;
	std::vector<int> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

	std::cout << reduce(data) << '\n';
	cudaCheck(hipDeviceSynchronize());
	cudaCheckLastError();
}