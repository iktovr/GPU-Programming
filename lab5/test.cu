#include "hip/hip_runtime.h"
#include <iostream>

#include "../common/error_checkers.hpp"

__global__ void kernel(int* a, int* b, int* c, int (*p)(int*, int*)) {
    (*c) = p(a, b);
}

__device__ int Min(int* a, int* b) {
    return (*a < *b) ? *a : *b;
}

int main() {
    int a, b, c;
    std::cin >> a >> b;

    int *dev_a, *dev_b, *dev_c;
    cudaCheck(hipMalloc(&dev_a, sizeof(int)));
    cudaCheck(hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&dev_b, sizeof(int)));
    cudaCheck(hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&dev_c, sizeof(int)));
    cudaCheck(hipMemcpy(dev_c, &c, sizeof(int), hipMemcpyHostToDevice));

    kernel<<<1, 32>>>(dev_a, dev_b, dev_c, &Min);
    cudaCheck(hipDeviceSynchronize());
    cudaCheckLastError();

    cudaCheck(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << c << '\n';
    cudaCheck(hipFree(dev_a));
    cudaCheck(hipFree(dev_b));
    cudaCheck(hipFree(dev_c));
}
