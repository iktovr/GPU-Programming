#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <limits> 

#include "../../common/error_checkers.hpp"
#include "../utils.hpp"
#include "../bitonic_sort.hpp"

// #define TIME
#ifdef TIME
#include "../../common/cuda_timer.hpp"
#endif

template <class T>
void bitonic_sort(std::vector<T>& data) {
    T *dev_data;
    cudaCheck(hipMalloc(&dev_data, sizeof(T) * data.size()));
    cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(T) * data.size(), hipMemcpyHostToDevice));

#ifdef TIME
    cudaStartTimer();
#endif

    bitonic_sort(dev_data, data.size(), std::numeric_limits<T>::max());

#ifdef TIME
    float t;
    cudaEndTimer(t);
    std::cout << t;
#endif

    cudaCheck(hipMemcpy(data.data(), dev_data, sizeof(T) * data.size(), hipMemcpyDeviceToHost));
    cudaCheck(hipFree(dev_data));
}

int main() {
	int n;
	std::cin >> n;
	std::vector<long long> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

    bitonic_sort(data);

    #ifndef TIME
    for (int i = 0; i < n; ++i) {
		std::cout << data[i] << ' ';
	}
    std::cout << '\n';
    #endif
    return 0;
}