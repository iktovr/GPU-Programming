#include <vector>
#include <iostream>

#include "../../common/error_checkers.hpp"
#include "../utils.hpp"
#include "../bitonic_sort.hpp"

std::vector<int> bitonic_sort(std::vector<int> data) {
    int *dev_data;
    cudaCheck(hipMalloc(&dev_data, sizeof(int) * data.size()));
    cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(int) * data.size(), hipMemcpyHostToDevice));

    bitonic_sort(dev_data, data.size(), 0x7FFFFFFF);

    cudaCheck(hipMemcpy(data.data(), dev_data, sizeof(int) * data.size(), hipMemcpyDeviceToHost));
    cudaCheck(hipFree(dev_data));
    return data;
}

int main() {
	int n;
	std::cin >> n;
	std::vector<int> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

    std::vector<int> res = bitonic_sort(data);
    for (int i = 0; i < n; ++i) {
		std::cout << res[i] << ' ';
	}
    std::cout << '\n';
}