#include <vector>
#include <iostream>

#include "../../common/error_checkers.hpp"
#include "../utils.hpp"
#include "../reduce.hpp"

template <class T> 
__device__ inline T add_func(T x, T y) {
    return x + y;
}

__device__ func_pointer<int> dev_add_func = add_func<int>;

int reduce(const std::vector<int>& data) {
	int *dev_data;
	cudaCheck(hipMalloc(&dev_data, sizeof(int) * data.size()));
	cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(int) * data.size(), hipMemcpyHostToDevice));

	func_pointer<int> h_add_func;
	cudaCheck(hipMemcpyFromSymbol(&h_add_func, HIP_SYMBOL(dev_add_func), sizeof(func_pointer<int)));

	int res = reduce(dev_data, data.size(), h_add_func, 0);
	cudaCheck(hipFree(dev_data));
	return res;
}

int main() {
	int n;
	std::cin >> n;
	std::vector<int> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

	std::cout << reduce(data) << '\n';
}