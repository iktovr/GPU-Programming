#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
// #include <algorithm>

#include "../../common/error_checkers.hpp"
#include "../hist.hpp"

template <class T>
T max(std::vector<T> v) {
    T m = v[0];
    for (T a: v) {
        if (a > m) {
            m = a;
        }
    }
    return m;
}

std::vector<int> histogram(const std::vector<int>& data) {
    int *dev_data, *dev_hist;
    cudaCheck(hipMalloc(&dev_data, sizeof(int) * data.size()));
    cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(int) * data.size(), hipMemcpyHostToDevice));

    int hist_size = max(data) + 1;
    dev_hist = histogram<int, int>(dev_data, data.size(), hist_size);

    std::vector<int> hist(hist_size);
    cudaCheck(hipMemcpy(hist.data(), dev_hist, sizeof(int) * hist_size, hipMemcpyDeviceToHost));
    cudaCheck(hipFree(dev_data));
    cudaCheck(hipFree(dev_hist));
    return hist;
}

int main() {
	int n;
	std::cin >> n;
	std::vector<int> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

    std::vector<int> res = histogram(data);
    for (size_t i = 0; i < res.size(); ++i) {
		std::cout << res[i] << ' ';
	}
    std::cout << '\n';
}