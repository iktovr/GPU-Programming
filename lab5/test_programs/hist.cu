#include <vector>
#include <iostream>
#include <algorithm>

#include "../../common/error_checkers.hpp"
#include "../hist.hpp"

std::vector<int> hist(const std::vector<int>& data) {
    int *dev_data, *dev_hist;
    cudaCheck(hipMalloc(&dev_data, sizeof(int) * data.size()));
    cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(int) * data.size(), hipMemcpyHostToDevice));

    int hist_size = *std::max_element(data.begin(), data.end()) + 1;
    dev_hist = hist(dev_data, data.size(), hist_size);

    std::vector<int> hist(hist_size);
    cudaCheck(hipMemcpy(hist.data(), dev_hist, sizeof(int) * hist_size, hipMemcpyDeviceToHost));
    cudaCheck(hipFree(dev_data));
    cudaCheck(hipFree(dev_hist));
    return hist;
}

int main() {
	int n;
	std::cin >> n;
	std::vector<int> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

    std::vector<int> res = hist(data);
    for (size_t i = 0; i < res.size(); ++i) {
		std::cout << res[i] << ' ';
	}
    std::cout << '\n';
}