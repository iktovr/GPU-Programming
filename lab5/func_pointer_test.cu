#include "hip/hip_runtime.h"
#include <iostream>

#include "../common/error_checkers.hpp"

template <class T>
using func_t = T (*) (T, T);

template <class T> 
__device__ inline T add_func (T x, T y)
{
    return x + y;
}

template <class T> 
__device__ inline T mul_func (T x, T y)
{
    return x * y;
}

// template <class T> 
__device__ func_t<int> p_add_func = add_func<int>;
// template <class T> 
// __device__ func_t<int> p_mul_func = mul_func<int>;

template <class T> 
__global__ void kernel(T* a, T* b, T* c, func_t<T> p) {
    (*c) = p(*a, *b);
}

int main() {
    int a, b, c;
    std::cin >> a >> b;

    func_t<int> h_add_func;
    int *dev_a, *dev_b, *dev_c;
    cudaCheck(hipMalloc(&dev_a, sizeof(int)));
    cudaCheck(hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&dev_b, sizeof(int)));
    cudaCheck(hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMalloc(&dev_c, sizeof(int)));
    cudaCheck(hipMemcpy(dev_c, &c, sizeof(int), hipMemcpyHostToDevice));

    cudaCheck(hipMemcpyFromSymbol(&h_add_func, HIP_SYMBOL(p_add_func), sizeof(func_t<int>)));

    kernel<int><<<1, 32>>>(dev_a, dev_b, dev_c, h_add_func);
    cudaCheck(hipDeviceSynchronize());
    cudaCheckLastError();

    cudaCheck(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << c << '\n';
    cudaCheck(hipFree(dev_a));
    cudaCheck(hipFree(dev_b));
    cudaCheck(hipFree(dev_c));
}
