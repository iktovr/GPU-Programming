#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "../common/error_checkers.hpp"

__global__ void hist(int *data, int size, int *hist) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    while (id < size) {
        atomicAdd(hist + data[id], 1);

        id += offset;
    }
}

std::vector<int> hist(const std::vector<int>& data, int hist_size) {
    int *dev_data, *dev_hist;
    cudaCheck(hipMalloc(&dev_data, sizeof(int) * data.size()));
    cudaCheck(hipMalloc(&dev_hist, sizeof(int) * hist_size));
    cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(int) * data.size(), hipMemcpyHostToDevice));
    cudaCheck(hipMemset(dev_hist, 0, sizeof(int) * hist_size));

    hist<<<1024, 1024>>>(dev_data, data.size(), dev_hist);
    cudaCheck(hipDeviceSynchronize());
    cudaCheckLastError();

    std::vector<int> hist(hist_size);
    cudaCheck(hipMemcpy(hist.data(), dev_hist, sizeof(int) * hist_size, hipMemcpyDeviceToHost));
    cudaCheck(hipFree(dev_data));
    cudaCheck(hipFree(dev_hist));
    return hist;
}

int main() {
	int n;
	std::cin >> n;
	std::vector<int> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

    std::vector<int> res = hist(data, 32);
    for (size_t i = 0; i < res.size(); ++i) {
		std::cout << res[i] << ' ';
	}
    std::cout << '\n';
}