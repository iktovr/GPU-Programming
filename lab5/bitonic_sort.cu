#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "../common/error_checkers.hpp"

const size_t BLOCK_SIZE = 1024;
const size_t BLOCK_COUNT = 1024;

__device__ void bitonic_merge(int i, int *data, int m) {
    int tmp, k;
    for (int b = m; b >= 2; b >>= 1) {
        if ((i & (b - 1)) < (b >> 1)) {
            k = i + (b >> 1);
            if (((i & m) && (data[i] < data[k])) || 
                (!(i & m) && (data[i] > data[k]))) {
                tmp = data[i];
                data[i] = data[k];
                data[k] = tmp;
            }
        }
        __syncthreads();
    }
}

__global__ void bitonic_sort_shared_memory(int *data, int size) {
    extern __shared__ int sdata[];

    int tid = threadIdx.x;
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
    int max_m = (size < BLOCK_SIZE) ? size : BLOCK_SIZE;

    while (id < size) {
        sdata[tid] = data[id];
		__syncthreads();

        for (int m = 2; m <= max_m; m <<= 1) {
            bitonic_merge(tid, sdata, m);
        }

        data[id] = sdata[tid];
		
        id += offset;
    }
}

__global__ void bitonic_sort_shared_memory(int *data, int size, int m) {
    extern __shared__ int sdata[];

    int tid = threadIdx.x;
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	while (id < size) {
		sdata[tid] = data[id];
		__syncthreads();

        bitonic_merge(tid, sdata, m);

        data[id] = sdata[tid];
		
        id += offset;
	}
}

__global__ void bitonic_sort_global_memory(int *data, int size, int m, int b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    int tmp, k;

    while (i < size) {
        if ((i & (b - 1)) < (b >> 1)) {
            k = i + (b >> 1);
            if (((i & m) && (data[i] < data[k])) || 
                (!(i & m) && (data[i] > data[k]))) {
                tmp = data[i];
                data[i] = data[k];
                data[k] = tmp;
            }
        }

        i += offset;
    }
}

template <class T>
T ceil_2_pow(T a) {
	while ((a & (a - 1)) != 0) {
		a &= a - 1;
	}
	return a << 1;
}

std::vector<int> bitonic_sort(std::vector<int> data) {
    size_t data_size = ceil_2_pow(data.size());
    int *dev_data;
    cudaCheck(hipMalloc(&dev_data, sizeof(int) * data.size()));
    cudaCheck(hipMemcpy(dev_data, data.data(), sizeof(int) * data.size(), hipMemcpyHostToDevice));

    bitonic_sort_shared_memory<<<BLOCK_COUNT, BLOCK_SIZE, sizeof(int) * BLOCK_SIZE>>>(dev_data, data.size());
    cudaCheck(hipDeviceSynchronize());
    cudaCheckLastError();
    for (size_t m = BLOCK_SIZE; m <= data.size(); m <<= 1) {
        for (size_t b = m; b > BLOCK_SIZE; b >>= 1) {
            bitonic_sort_global_memory<<<BLOCK_COUNT, BLOCK_SIZE>>>(dev_data, data.size(), m, b);
            cudaCheck(hipDeviceSynchronize());
            cudaCheckLastError();
        }
        bitonic_sort_shared_memory<<<BLOCK_COUNT, BLOCK_SIZE, sizeof(int) * BLOCK_COUNT>>>(dev_data, data.size(), m);
        cudaCheck(hipDeviceSynchronize());
        cudaCheckLastError();
    }

    cudaCheck(hipMemcpy(data.data(), dev_data, sizeof(int) * data.size(), hipMemcpyDeviceToHost));
    cudaCheck(hipFree(dev_data));
    return data;
}

int main() {
	int n;
	std::cin >> n;
	std::vector<int> data(n);
	for (int i = 0; i < n; ++i) {
		std::cin >> data[i];
	}

    std::vector<int> res = bitonic_sort(data);
    for (int i = 0; i < n; ++i) {
		std::cout << res[i] << ' ';
	}
    std::cout << '\n';
}